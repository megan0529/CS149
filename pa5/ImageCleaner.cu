#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#define SIZEX    1024
#endif
#ifndef SIZEY
#define SIZEY    1024
#endif
#define BLOCK_SIZE 16
#define PI 3.14159265

typedef struct {
	int size;
	int stride;
	float *elements;
} Matrix;
//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------

//row and col is within the sub-matrix
__device__ float getElement(const Matrix A, int row, int col) {
	return A.elements[row * A.stride + col];
}

__device__ void setElement(Matrix A, int row, int col, float value) {
	A.elements[row * A.stride + col] = value;
}

__device__ Matrix getSubMatrix(Matrix A, int blockRow, int blockCol) {
	Matrix Asub;
	Asub.size = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = A.elements + blockRow * BLOCK_SIZE * A.stride + blockCol * BLOCK_SIZE;
	return Asub;
}

__global__ void FT_Kernel(const Matrix realMatrix, const Matrix imagMatrix, const Matrix sinMatrix,
		const Matrix cosMatrix, Matrix resultRealMatrix, Matrix resultImagMatrix) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
	// the sub-matrix block this thread block is responsible for
	Matrix subResultReal = getSubMatrix(resultRealMatrix, blockCol, blockRow);
	Matrix subResultImag = getSubMatrix(resultImagMatrix, blockCol, blockRow);
	//the sub-matrix entry this thread is responsible for
	int row = threadIdx.y;
	int col = threadIdx.x;

	float Creal = 0;
	float Cimag = 0;

	for (int m = 0; m < (realMatrix.size / BLOCK_SIZE); ++m) {
		Matrix realSub = getSubMatrix(realMatrix, blockRow, m);
		Matrix imagSub = getSubMatrix(imagMatrix, blockRow, m);
		Matrix sinSub = getSubMatrix(sinMatrix, m, blockCol);
		Matrix cosSub = getSubMatrix(cosMatrix, m, blockCol);

		__shared__
		float realShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float imagShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float sinShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float cosShare[BLOCK_SIZE][BLOCK_SIZE];

		realShare[row][col] = getElement(realSub, row, col);
		imagShare[row][col] = getElement(imagSub, row, col);
		sinShare[row][col] = getElement(sinSub, row, col);
		cosShare[row][col] = getElement(cosSub, row, col);

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; ++i) {
			Creal += realShare[row][i] * cosShare[i][col] + imagShare[row][i] * sinShare[i][col];
			Cimag += realShare[row][i] * sinShare[i][col]*(-1) + imagShare[row][i] * cosShare[i][col];
		}
		__syncthreads();
	}
	//Write sub-result to device memory
	//each thread writes one element
	//must write the result to a buffer instead of write back, since other blocks might have not finished.
	setElement(subResultReal, col, row, Creal);
	setElement(subResultImag, col, row, Cimag);

}

__global__ void iFT_Kernel(const Matrix realMatrix, const Matrix imagMatrix, const Matrix sinMatrix,
		const Matrix cosMatrix, Matrix resultRealMatrix, Matrix resultImagMatrix) {
	//blockIdx and threadIdx are there as keyword
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
	// the sub-matrix block this thread block is responsible for
	Matrix subResultReal = getSubMatrix(resultRealMatrix, blockCol, blockRow);
	Matrix subResultImag = getSubMatrix(resultImagMatrix, blockCol, blockRow);
	//the sub-matrix entry this thread is responsible for
	int row = threadIdx.y;
	int col = threadIdx.x;

	float Creal = 0;
	float Cimag = 0;

	for (int m = 0; m < (realMatrix.size / BLOCK_SIZE); ++m) {
		Matrix realSub = getSubMatrix(realMatrix, blockRow, m);
		Matrix imagSub = getSubMatrix(imagMatrix, blockRow, m);
		Matrix sinSub = getSubMatrix(sinMatrix, m, blockCol);
		Matrix cosSub = getSubMatrix(cosMatrix, m, blockCol);

		__shared__
		float realShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float imagShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float sinShare[BLOCK_SIZE][BLOCK_SIZE];
		__shared__
		float cosShare[BLOCK_SIZE][BLOCK_SIZE];

		realShare[row][col] = getElement(realSub, row, col);
		imagShare[row][col] = getElement(imagSub, row, col);
		sinShare[row][col] = getElement(sinSub, row, col);
		cosShare[row][col] = getElement(cosSub, row, col);

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; ++i) {
			Creal += realShare[row][i] * cosShare[i][col] - imagShare[row][i] * sinShare[i][col];
			Cimag += imagShare[row][i] * cosShare[i][col] + realShare[row][i] * sinShare[i][col];
		}
		__syncthreads();
	}
	//Write sub-result to device memory
	//each thread writes one element
	setElement(subResultReal, col, row, Creal / SIZEX);
	setElement(subResultImag, col, row, Cimag / SIZEX);

}

__global__ void sincosKernel(Matrix sinMatrix, Matrix cosMatrix) {
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	float angel = row * col * 2 * 3.1415926536 / sinMatrix.size;
	//each thread is responsible for filling in only one entry
	sinMatrix.elements[row * sinMatrix.size + col] = __sinf(angel);
	cosMatrix.elements[row * cosMatrix.size + col] = __cosf(angel);

}

__global__ void filterKernel(Matrix realMatrix, Matrix imagMatrix) {
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int low8 = realMatrix.size / 8;
	int high8 = realMatrix.size - low8;
	if (!((row < low8 && col < low8) || (row < low8 && col >= high8) || (row >= high8 && col >= high8)
			|| (row >= high8 && col < low8))) {
		realMatrix.elements[row * realMatrix.size + col] = 0;
		imagMatrix.elements[row * imagMatrix.size + col] = 0;
	}
}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y) {
// check that the sizes match up
	assert(size_x == SIZEX);
	assert(size_y == SIZEY);

	int matSize = size_x * size_y * sizeof(float);

// These variables are for timing purposes
	float transferDown = 0, transferUp = 0, execution = 0;
	hipEvent_t start, stop;

	CUDA_ERROR_CHECK(hipEventCreate(&start));
	CUDA_ERROR_CHECK(hipEventCreate(&stop));

// Create a stream and initialize it
	hipStream_t filterStream;
	CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

// Alloc space on the device
	Matrix realMatrix, imagMatrix, tmpRealMatrix, tmpImagMatrix, sinMatrix, cosMatrix;
	realMatrix.size = imagMatrix.size = tmpRealMatrix.size = tmpImagMatrix.size = sinMatrix.size = cosMatrix.size =
			size_x;
	realMatrix.stride = imagMatrix.stride = tmpRealMatrix.stride = tmpImagMatrix.stride = sinMatrix.stride =
			cosMatrix.stride = size_x;
	//device memory could be allocated in the host, but not dereference in the host
	CUDA_ERROR_CHECK(hipMalloc((void** )&realMatrix.elements, matSize));
	CUDA_ERROR_CHECK(hipMalloc((void** )&imagMatrix.elements, matSize));

	CUDA_ERROR_CHECK(hipMalloc((void** )&tmpRealMatrix.elements, matSize));
	CUDA_ERROR_CHECK(hipMalloc((void** )&tmpImagMatrix.elements, matSize));

	CUDA_ERROR_CHECK(hipMalloc((void** )&sinMatrix.elements, matSize));
	CUDA_ERROR_CHECK(hipMalloc((void** )&cosMatrix.elements, matSize));

// Start timing for transfer down
	CUDA_ERROR_CHECK(hipEventRecord(start, filterStream));

// Here is where we copy matrices down to the device
	CUDA_ERROR_CHECK(hipMemcpy(realMatrix.elements, real_image, matSize, hipMemcpyHostToDevice));
	CUDA_ERROR_CHECK(hipMemcpy(imagMatrix.elements, imag_image, matSize, hipMemcpyHostToDevice));

// Stop timing for transfer down
	CUDA_ERROR_CHECK(hipEventRecord(stop, filterStream));
	CUDA_ERROR_CHECK(hipEventSynchronize(stop));
	CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown, start, stop));

// Start timing for the execution
	CUDA_ERROR_CHECK(hipEventRecord(start, filterStream));

//----------------------------------------------------------------
// TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
//        HERE BETWEEN THE CALLS FOR STARTING AND
//        FINISHING TIMING FOR THE EXECUTION PHASE
// BEGIN ADD KERNEL CALLS
//----------------------------------------------------------------

// This is an example kernel call, you should feel free to create
// as many kernel calls as you feel are needed for your program
// Each of the parameters are as follows:
//    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
//    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
//    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
//    4. Stream to execute kernel on, should always be 'filterStream'
//
// Also note that you pass the pointers to the device memory to the kernel call
	dim3 dimGrid(size_x / BLOCK_SIZE, size_y / BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

sincosKernel<<<dimGrid, dimBlock, 0, filterStream>>> (sinMatrix, cosMatrix);
	FT_Kernel<<<dimGrid, dimBlock, 0, filterStream>>> (realMatrix, imagMatrix, sinMatrix, cosMatrix, tmpRealMatrix, tmpImagMatrix);
	FT_Kernel<<<dimGrid, dimBlock, 0, filterStream>>> (tmpRealMatrix, tmpImagMatrix, sinMatrix, cosMatrix, realMatrix, imagMatrix);
	filterKernel<<<dimGrid, dimBlock, 0, filterStream>>> (realMatrix, imagMatrix);
	iFT_Kernel<<<dimGrid, dimBlock, 0, filterStream>>> (realMatrix, imagMatrix, sinMatrix, cosMatrix, tmpRealMatrix, tmpImagMatrix);
	iFT_Kernel<<<dimGrid, dimBlock, 0, filterStream>>> (tmpRealMatrix, tmpImagMatrix, sinMatrix, cosMatrix, realMatrix, imagMatrix);


  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
	CUDA_ERROR_CHECK(hipEventRecord(stop, filterStream));
	CUDA_ERROR_CHECK(hipEventSynchronize(stop));
	CUDA_ERROR_CHECK(hipEventElapsedTime(&execution, start, stop));

// Start timing for the transfer up
	CUDA_ERROR_CHECK(hipEventRecord(start, filterStream));

// Here is where we copy matrices back from the device
	CUDA_ERROR_CHECK(hipMemcpy(real_image, realMatrix.elements, matSize, hipMemcpyDeviceToHost));
	CUDA_ERROR_CHECK(hipMemcpy(imag_image, imagMatrix.elements, matSize, hipMemcpyDeviceToHost));

// Finish timing for transfer up
	CUDA_ERROR_CHECK(hipEventRecord(stop, filterStream));
	CUDA_ERROR_CHECK(hipEventSynchronize(stop));
	CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp, start, stop));

// Synchronize the stream
	CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
// Destroy the stream
	CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
// Destroy the events
	CUDA_ERROR_CHECK(hipEventDestroy(start));
	CUDA_ERROR_CHECK(hipEventDestroy(stop));

// Free the memory
	CUDA_ERROR_CHECK(hipFree(realMatrix.elements));
	CUDA_ERROR_CHECK(hipFree(imagMatrix.elements));
	CUDA_ERROR_CHECK(hipFree(tmpRealMatrix.elements));
	CUDA_ERROR_CHECK(hipFree(tmpImagMatrix.elements));
	CUDA_ERROR_CHECK(hipFree(sinMatrix.elements));
	CUDA_ERROR_CHECK(hipFree(cosMatrix.elements));

// Dump some usage statistics
	printf("CUDA IMPLEMENTATION STATISTICS:\n");
	printf("  Host to Device Transfer Time: %f ms\n", transferDown);
	printf("  Kernel(s) Execution Time: %f ms\n", execution);
	printf("  Device to Host Transfer Time: %f ms\n", transferUp);
	float totalTime = transferDown + execution + transferUp;
	printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
// Return the total time to transfer and execute
	return totalTime;
}

